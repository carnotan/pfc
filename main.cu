#include "hip/hip_runtime.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <float.h>
#include <thrust/remove.h>
#include <thrust/execution_policy.h>
#include <fstream>
#include "data_structures.h"
#include "reduce.h"
#include "auxiliares.h"
#include "mutation.h"
#include "crossover.h"
#include "fitness.h"
#include "selection.h"
#include "generation.h"
#include "replacement.h"
#include "debug.h"
#include <iostream>

//SHUFFLE
#include <algorithm>    // std::shuffle
#include <random>       // std::default_random_engine
#include <chrono>       // std::chrono::system_clock

#define THRESHOLD 0.012f 
#define PLANE_REGION 0.5f 
#define EPSILON FLT_EPSILON 
#define POPULATION_SIZE 200
#define CLOUD_SIZE 100000
#define MUTATION_D_INDEX 100 
#define MUTATION_RATE 0.25f
#define CROSS_D_INDEX 2.0f
#define CROSS_PROB 0.5f
#define GENE_SIZE 4.0f
#define MAX_EXECUTIONS 5000
#define SIGNIFICANT_PLANE 50
#define TOUR_SIZE 5
#define MAX_FAIL 3

/**
 * Función que executa o bucle principal do algoritmo. Ao contrario dos 
 * algoritmos xenéticos clásicos non atopa unha solución, senón que vai atopando
 * unha "parte" da solución . O criterio de parada do algoritmo é que todos os 
 * puntos da nube estén asignados a un plano (ou non queden máis planos) . O 
 * criterio de converxencia para a poboación é que todos os individuos da
 * poboación teñan o mesmo número de puntos axustados dentro deles.  
 * 
 * @param pop_size Tamaño da poboación.
 * @param cloud_size Tamaño da nube.
 * @param t Umbral do plano.
 * @param r Rexión do plano.
 * @param max_exec Número máximo de execucións permitida.  
 * @param max_fail Número máximo de veces que se permite converxer ao algoritmo 
 * sen atopar un plano significativo.
 * @param population Punteiro a host_vector<Solution> que contén a poboación.
 * @param mating_pool Punteiro a host_vector<Solution> que contén a piscina de 
 * reprodución.
 * @param h_cloud Punteiro a host_vector<Point> que contén a nube en memoria 
 * principal do sistema.
 * @param d_cloud Punteiro a device_vector<Point> que contén a nube en memoria 
 * global de dispositivo.
 * @param plane_min_size Tamaño mínimo dun plano para consideralo 
 * arquitectónicamente significativo.
 * @param upper Punteiro ao array de límites superiores.
 * @param lower Punteiro ao array de límites inferiores.
 * @param tour_size Tamaño do torneo.
 * @param mutation_rate Probabilidade de mutación.
 * @param mutation_index Índice da función de distribución de probabilidade 
 * polinomial do operador de mutación.
 * @param cross_rate Probabilidade de cruce.
 * @param cross_index Índice da función de distribución de probabilidade do 
 * operador de cruce.
 * @param fast_convergence Booleano para indicar se se desexa activar a mellora
 * de converxencia anticipada.
 * @param min_growth Crecemento mínimo relativo aceptable no modo de 
 * converxencia anticipada. 
 * @param max_block_size Tamaño máximo dos bloques para os kernels de CUDA.
 * @param max_grid_size Tamaño máximo das mallas para os kernels de CUDA.
 * @return 0 se o algorimto se executa correctamente, código de erro noutro 
 * caso.
 */
int bucle(size_t pop_size, size_t *cloud_size, float t, float r,
        int max_exec, int max_fail, thrust::host_vector<Solution> *population,
        thrust::host_vector<Solution> *mating_pool,
        thrust::host_vector<Point> *h_cloud,
        thrust::device_vector <Point> *d_cloud, size_t plane_min_size,
        float *upper, float *lower, int tour_size, float mutation_rate,
        float mutation_index, float cross_rate, float cross_index,
        bool fast_convergence, float min_growth, int max_block_size,
        int max_grid_size) {

    int planes_found = 1;
    float av_fitness;
    float time_acumulated = 0;
    std::ofstream myfile;
    myfile.open("performance.txt", std::ios::out | std::ios::trunc);
    hipEvent_t start, stop, eval_start, eval_stop;
    float miliseconds;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&eval_start);
    hipEventCreate(&eval_stop);
    hipDeviceProp_t prop;
    int device;
    gpuErrchk(hipGetDevice(&device));
    gpuErrchk(hipGetDeviceProperties(&prop, device));

    //cambio de criterio de convergencia
    float * average = (float*) malloc(sizeof (float));
    *average = 1;
    int * not_improved = (int *) malloc(sizeof (int));

    int fail = 0;
    int exec_count = 0;
    int result;
    float *mut_rate = (float*) malloc(sizeof (float));
    if (mut_rate == NULL) {
        printf("Non hai memoria suficiente para facer malloc\n");
        printf("Erro dentro do bucle\n");
        return (-1);
    }
    float *mut_d_index = (float*) malloc(sizeof (float));
    if (mut_d_index == NULL) {
        printf("Non hai memoria suficiente para facer malloc\n");
        printf("Erro dentro do bucle\n");
        return (-1);
    }
    Point *p_d_cloud = thrust::raw_pointer_cast(d_cloud->data());
    thrust::device_vector<int> fitted(*cloud_size);
    thrust::device_vector<int> region(*cloud_size);
    thrust::device_vector<float>fit(*cloud_size);
    myfile << "Inicio do algoritmo." << std::endl;
    myfile << "Tamaño da nube: " << *cloud_size << std::endl;
    myfile << "Tamaño da poboación: " << pop_size << std::endl;
    myfile << "Tamaño máximo de bloque (CUDA): " << max_block_size << std::endl;
    myfile << "Tamaño máximo de malla (CUDA): " << max_grid_size << std::endl;
    //Criterio de parada do algoritmo
    while (*cloud_size > 0 && fail < max_fail) {
        hipEventRecord(start);
        *mut_rate = mutation_rate;
        *mut_d_index = mutation_index;
        exec_count = 0;
        thrust::fill(thrust::device, fitted.begin(), fitted.end(), 0);
        thrust::fill(thrust::device, region.begin(), region.end(), 0);
        thrust::fill(thrust::device, fit.begin(), fit.end(), 0.f);
        if ((result = generate_population(pop_size, *cloud_size, population,
                h_cloud)) != 0) //generation.cu
            return result;
        myfile << "#################################" << std::endl;
        myfile << "Plano número " << planes_found << std::endl;
        normalize(population, pop_size); //generation.cu
        evaluate_population_cuda(t, r, EPSILON,
                p_d_cloud, population, *cloud_size, pop_size, &fitted, &region,
                &fit, max_block_size, max_grid_size, device, prop); //fitness.cu
        //Criterio de converxencia.
        while (!is_converged(*population, pop_size, average, not_improved,
                fast_convergence, min_growth)) {//auxiliares.cu
            if ((result = tournament_selection(*population, mating_pool,
                    pop_size, tour_size)) != 0) //selection.cu
                return -3;
            if ((result = recombine(mating_pool, pop_size, cross_rate,
                    cross_index, upper, lower, EPSILON)) != 0) //crossover.cu
                return -4;
            mutation(pop_size, mating_pool, mut_rate, mut_d_index, upper,
                    lower); //mutation.cu 
            hipEventRecord(eval_start);
            evaluate_population_cuda(t, r, EPSILON, p_d_cloud, mating_pool,
                    *cloud_size, pop_size, &fitted, &region, &fit,
                    max_block_size, max_grid_size, device, prop); //fitness.cu
            hipEventRecord(eval_stop);
            hipEventSynchronize(eval_stop);
            hipEventElapsedTime(&miliseconds, eval_start, eval_stop);
            time_acumulated += miliseconds;
            myfile << "Tempo para avaliar a poboación, ciclo " << exec_count
                    << ": " << miliseconds << "ms" << std::endl;
            replacement(population, *mating_pool, pop_size); //replacement.cu
            av_fitness = average_fitness(*population, pop_size);
            myfile << "Fitness medio da poboación: " << av_fitness << std::endl;
            exec_count++;
            recalculate_parameters(mut_d_index, mut_rate, GENE_SIZE, exec_count,
                    max_exec, mutation_index); //mutation.cu
        }

        //Unha vez converxeu a poboación, retiramos da nube os puntos
        //pertencentes ao plano atopado.
        if (population->operator[](0).points_fitted > plane_min_size) {
            evaluate_population_cuda(t, r, EPSILON, p_d_cloud,
                    population, *cloud_size, 1, &fitted, &region,
                    &fit, max_block_size, max_grid_size, device,
                    prop); //fitness.cu
            thrust::detail::normal_iterator<thrust::device_ptr<Point> > new_end
                    = thrust::remove_if(thrust::device, d_cloud->begin(),
                    d_cloud->end(), fitted.begin(), thrust::identity<int>());
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&miliseconds, start, stop);
            myfile << "Tempo de execución total para atopar o plano "
                    << planes_found << ": " << miliseconds << " ms" <<
                    std::endl;
            myfile << "Tempo de execución de avaliación de fitnes para este"
                    " plano: " << time_acumulated << " ms ." << std::endl;
            myfile << "Tempo medio de avaliación por ciclo: " <<
                    time_acumulated / exec_count << " ms." << std::endl;
            myfile << "Porcentaxe de tempo respecto do tempo total: " <<
                    time_acumulated / miliseconds * 100 << " %" << std::endl;
            myfile << "#################################" << std::endl;
            time_acumulated = 0;
            write_solution(population->operator[](0)); //auxiliares.cu
            *cloud_size = *cloud_size - population->operator[](0).points_fitted;
            fail = 0;
            thrust::copy(d_cloud->begin(), new_end, h_cloud->begin());
            exec_count = 0;
            planes_found++;

        } else {
            time_acumulated=0;
            exec_count=0;
            fail++;
        }
    }
    myfile.close();
    return 0;

}

/**
 * Función auxiliar que crea a nube de puntos sintética. Engade un delta 
 * aleatorio a cada punto para xerar ruído na nube.
 * 
 * @param cloud Punteiro a host_vector<Point> que contén a nube na memoria do 
 * sistema.
 * @param cloud_size Tamaño da nube
 * @param s  Estructura Solution cos parámetros do plano que vamos a empregar 
 * para xerar puntos
 * @param size Número de puntos a xerar.
 * @param offset Punto da nube onde empezamos a escribir.
 */
void generate_cloud(thrust::host_vector<Point> *cloud, Solution s, size_t size,
        size_t offset) {

    float z = 0;
    float delta = 0;
    for (int i = 0; i < 100; i++) {
        for (int j = 0; j < size / 100; j++) {
            z = ((-s.chromosome[3])+(-s.chromosome[0] * i)+
                    (-s.chromosome[1] * j)) / (s.chromosome[2]);
            delta = ((2 * ((float) rand() / (float) RAND_MAX)) - 1) / 200.0f;

            cloud->operator[](i * size / 100 + j + offset).coordinates[0] =
                    (float) i + delta;
            cloud->operator[](i * size / 100 + j + offset).coordinates[1] =
                    (float) j + delta;
            cloud->operator[](i * size / 100 + j + offset).coordinates[2] =
                    (float) z + delta;
        }
    }

}

int main(int argc, char **argv) {
    srand(time(NULL));
    size_t * cloud_size = (size_t *) malloc(sizeof (size_t));
    size_t pop_size;
    size_t offset = 0;
    size_t size;
    float mut_rate;
    float mut_d_index;
    float cross_d_index;
    float cross_rate;
    float r;
    float t;
    int max_exec;
    int significant_plane;
    int tour_size;
    int max_fail;
    bool fast_convergence;
    float min_growth = 0.f;
    float *upper = (float *) malloc(4 * sizeof (float));
    float *lower = (float *) malloc(4 * sizeof (float));
    int max_block_size;
    int max_grid_size;
    upper[0] = 1.0f;
    upper[1] = 1.0f;
    upper[2] = 1.0f;
    upper[3] = 1.0f;
    lower[0] = -1.0f;
    lower[1] = -1.0f;
    lower[2] = -1.0f;
    lower[3] = -1.0f;
    Solution s;


    switch (argc) {
        case 1: std::cout << "Modo normal" << std::endl;
            *cloud_size = CLOUD_SIZE;
            pop_size = POPULATION_SIZE;
            mut_rate = MUTATION_RATE;
            mut_d_index = MUTATION_D_INDEX;
            cross_d_index = CROSS_D_INDEX;
            cross_rate = CROSS_PROB;
            r = PLANE_REGION;
            t = THRESHOLD;
            max_exec = MAX_EXECUTIONS;
            significant_plane = SIGNIFICANT_PLANE;
            tour_size = TOUR_SIZE;
            max_fail = MAX_FAIL;
            fast_convergence = false;
            max_block_size = 256;
            max_grid_size = 128;
            break;
        case 7: std::cout << "Modo rendemento" << std::endl;
            if ((pop_size = std::stoi(argv[1])) % 2) {
                std::cout << "A poboación non pode ser impar" << std::endl;
                exit(-1);
            }
            if ((*cloud_size = std::stoi(argv[2])) % 10000) {
                std::cout << "O tamaño da nube de puntos debe ser múltiplo"
                        " de 10000" << std::endl;
                exit(-1);
            }
            if (std::stoi(argv[3])) {
                fast_convergence = true;
            } else
                fast_convergence = false;
            min_growth = std::stof(argv[4]);
            max_block_size = std::stoi(argv[5]);
            if (!isPow2(max_block_size) || max_block_size > 1024) {
                std::cout << "Tamaño de bloque non válido!!" << std::endl;
                exit(-1);
            }
            max_grid_size = std::stoi(argv[6]);
            if (!isPow2(max_grid_size) || max_grid_size > 65535) {
                std::cout << "Tamaño de malla non válido!!" << std::endl;
                exit(-1);
            }
            mut_rate = MUTATION_RATE;
            mut_d_index = MUTATION_D_INDEX;
            cross_d_index = CROSS_D_INDEX;
            cross_rate = CROSS_PROB;
            r = PLANE_REGION;
            t = THRESHOLD;
            max_exec = MAX_EXECUTIONS;
            significant_plane = SIGNIFICANT_PLANE;
            tour_size = TOUR_SIZE;
            max_fail = MAX_FAIL;
            break;
        case 17: std::cout << "Modo avanzado" << std::endl;
            if ((pop_size = std::stoi(argv[1])) % 2) {
                std::cout << "A poboación non pode ser impar" << std::endl;
                exit(-1);
            }
            if ((*cloud_size = std::stoi(argv[2])) % 10000) {
                std::cout << "O tamaño da nube de puntos debe ser múltiplo "
                        "de 10000" << std::endl;
                exit(-1);
            }
            r = std::stof(argv[3]);
            t = std::stof(argv[4]);
            if (r < t) {
                std::cout << "O parámetro r non pode ser menor que o parámetro "
                        "t" << std::endl;
                exit(-1);
            }
            mut_d_index = std::stof(argv[5]);
            if ((mut_rate = std::stof(argv[6])) > 1.0f) {
                std::cout << "A probabilidade de mutación non pode ser maior "
                        "ca 1!!" << std::endl;
                exit(-1);
            }
            cross_d_index = std::stof(argv[7]);

            if ((cross_rate = std::stof(argv[8])) > 1.0f) {
                std::cout << "A probabilidade de cruzamento non pode ser maior"
                        " ca 1!!" << std::endl;
                exit(-1);
            }
            max_exec = std::stoi(argv[9]);
            significant_plane = std::stoi(argv[10]);
            tour_size = std::stoi(argv[11]);
            max_fail = std::stoi(argv[12]);
            if (std::stoi(argv[13])) {
                fast_convergence = true;
            } else
                fast_convergence = false;
            min_growth = std::stof(argv[14]);
            max_block_size = std::stoi(argv[15]);
            if (!isPow2(max_block_size) || max_block_size > 1024) {
                std::cout << "Tamaño de bloque non válido!!" << std::endl;
                exit(-1);
            }
            max_grid_size = std::stoi(argv[16]);
            if (!isPow2(max_grid_size) || max_grid_size > 65535) {
                std::cout << "Tamaño de malla non válido!!" << std::endl;
                exit(-1);
            }
            break;
        default: std::cout << "Número de parámetros incorrecto" << std::endl;
            std::cout << "Uso: " << std::endl;
            std::cout << "Modo normal: Execución cos parámetros preestablecidos"
                    << std::endl;
            std::cout << "./paralelo" << std::endl;
            std::cout << "##################################################"
                    "####"
                    << std::endl;
            std::cout << "Modo rendemento: Estuda o rendemento do algoritmo "
                    "cambiando os parámetros que afectan ao tempo de execución"
                    << std::endl;
            std::cout << "./paralelo pop_size cloud_size fast_convergence "
                    "min_growth max_block_size max_grid_size" << std::endl;
            std::cout << "\tpop_size: tamaño da poboación (número par)"
                    << std::endl;
            std::cout << "\tcloud_size: tamaño da nube (múltiplo de 10000)"
                    << std::endl;
            std::cout << "\tfast_convergence 1 para activar a converxencia "
                    "adiantada, 0 para desactivala" << std::endl;
            std::cout << "\tmin_growth crecemento mínimo aceptable . "
                    "Recomendado"
                    "[0.00001,0.001] . (Con fast_convergence=0, introducir "
                    "calquera valor." << std::endl;
            std::cout << "\tmax_block_size: Tamaño de bloque dos kernels "
                    "(múltiplo de 2, menor de 1024)" << std::endl;
            std::cout << "\tmax_grid_size: Tamaño máximo da malla (múltiplo de"
                    " 2, menor de 65535" << std::endl;
            std::cout << "############################################"
                    "##########"
                    << std::endl;
            std::cout << "Modo avanzado:" << std::endl;
            std::cout << "./paralelo pop_size cloud_size r t mut_d_index mut_rate"
                    "cross_d_index cross_rate max_exec significant_plane "
                    "tour_size max_fail fast_convergence min_growth "
                    "max_block_size max_grid_size" << std::endl;
            exit(-1);
            break;
    }

    thrust::host_vector<Point> h_cloud(*cloud_size);
    thrust::host_vector<Solution> population(pop_size);
    thrust::host_vector<Solution> mating_pool(pop_size);

    s.chromosome[0] = 11;
    s.chromosome[1] = 16;
    s.chromosome[2] = 14;
    s.chromosome[3] = -15;
    size = (*cloud_size / 100)*20;

    generate_cloud(&h_cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = 8;
    s.chromosome[1] = 2;
    s.chromosome[2] = 3;
    s.chromosome[3] = 1;
    size = (*cloud_size / 100)*15;
    generate_cloud(&h_cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = -1;
    s.chromosome[1] = 4.5;
    s.chromosome[2] = 2.5;
    s.chromosome[3] = 12;
    size = (*cloud_size / 100)*12;
    generate_cloud(&h_cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = 18;
    s.chromosome[1] = 22;
    s.chromosome[2] = 1;
    s.chromosome[3] = 9;
    size = (*cloud_size / 100)*10;
    generate_cloud(&h_cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = -8;
    s.chromosome[1] = 0;
    s.chromosome[2] = 4;
    s.chromosome[3] = -3.25;
    size = (*cloud_size / 100)*10;
    generate_cloud(&h_cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = -5.5;
    s.chromosome[1] = 12;
    s.chromosome[2] = -1;
    s.chromosome[3] = -1;
    size = (*cloud_size / 100)*9;
    generate_cloud(&h_cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = 1;
    s.chromosome[1] = 1;
    s.chromosome[2] = 4;
    s.chromosome[3] = -5;
    size = (*cloud_size / 100)*8;
    generate_cloud(&h_cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = 12;
    s.chromosome[1] = 3;
    s.chromosome[2] = 5.5;
    s.chromosome[3] = -1;
    size = (*cloud_size / 100)*7;
    generate_cloud(&h_cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = -1;
    s.chromosome[1] = 8;
    s.chromosome[2] = 2;
    s.chromosome[3] = -9;
    size = (*cloud_size / 100)*6;
    generate_cloud(&h_cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = -3;
    s.chromosome[1] = -1;
    s.chromosome[2] = 8;
    s.chromosome[3] = -4;
    size = (*cloud_size / 100)*3;
    generate_cloud(&h_cloud, s, size, offset);
    offset += size;

    //Baraxamos aleatoriamente a poboación para enfrontar ao algoritmo ao peor 
    //caso posible (incrementa a diverxencia dentro dos warp no kernel 
    //de fitness)
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    shuffle(h_cloud.begin(), h_cloud.end(), std::default_random_engine(seed));
    thrust::device_vector<Point> d_cloud = h_cloud;


    int result = bucle(pop_size, cloud_size, t, r, max_exec, max_fail,
            &population, &mating_pool, &h_cloud, &d_cloud, significant_plane,
            upper, lower, tour_size, mut_rate, mut_d_index, cross_rate,
            cross_d_index, fast_convergence, min_growth, max_block_size,
            max_grid_size);

    if (result == 1) {
        hipDeviceReset();
        exit(0);
    }
    hipDeviceReset();
    exit(result);
}
