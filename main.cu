#include "hip/hip_runtime.h"

#include "data_structures.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "auxiliares.h"
#include "crossover.h"
#include "debug.h"
#include "generation.h"
#include "mutation.h"
#include "selection.h"
#include "fitness.h"
#include "replacement.h"
#include "eliminate.h"
#include <float.h>

#define THRESHOLD 0.015f 
#define PLANE_REGION 0.5f
#define EPSILON FLT_EPSILON
#define POPULATION_SIZE 200
#define CLOUD_SIZE 352600
#define MUTATION_D_INDEX 100
#define MUTATION_RATE 0.25f
#define CROSS_D_INDEX 2.0f
#define CROSS_PROB 0.5f
#define GENE_SIZE 4.0f
#define MAX_EXECUTIONS 5000
#define SIGNIFICANT_PLANE 50
#define TOUR_SIZE 5
#define MAX_FAIL 3

/**
 * Función que executa o bucle principal do algoritmo. Ao contrario dos 
 * algoritmos xenéticos clásicos non atopa unha solución, senón que vai atopando
 * unha "parte" da solución . O criterio de parada do algoritmo é que todos os 
 * puntos da nube estén asignados a un plano (ou non queden máis planos) . O 
 * criterio de converxencia para a poboación é que todos os individuos da
 * poboación teñan o mesmo número de puntos axustados dentro deles.
 * 
 * @param pop_size Tamaño da poboación.
 * @param cloud_size Tamaño da nube.
 * @param max_fail Número máximo de veces que se permite converxer ao algoritmo 
 * sen atopar un plano significativo.
 * @param population Punteiro a host_vector<Solution> que contén a poboación.
 * @param mating_pool Punteiro a host_vector<Solution> que contén a piscina de 
 * reprodución.
 * @param cloud  Punteiro a host_vector<Point> que contén a nube en memoria 
 * principal do sistema.
 * @param plane_min_size Tamaño mínimo dun plano para consideralo 
 * arquitectónicamente significativo.
 * @param upper Punteiro ao array de límites superiores.
 * @param lower Punteiro ao array de límites inferiores.
 * @return 0 se o bucle se executa correctamente, o código de erro noutro caso.
 */
int bucle(size_t pop_size, size_t *cloud_size, int max_fail,
        thrust::host_vector <Solution> *population,
        thrust::host_vector<Solution> *mating_pool,
        thrust::host_vector<Point> *cloud, size_t plane_min_size, float *upper,
        float *lower) {
    write_cloud(*cloud, *cloud_size, "nube.txt");

    int fail = 0;
    int result;
    int exec_count = 0;
    float *base = (float*) malloc(sizeof (float));
    float *mut_rate = (float*) malloc(sizeof (float));
    float *mut_d_index = (float*) malloc(sizeof (float));
    //Criterio de parada do algoritmo.
    while (*cloud_size > 0 && fail < max_fail) {
        *mut_rate = MUTATION_RATE;
        *mut_d_index = MUTATION_D_INDEX;
        exec_count = 0;
        if ((result = generate_population(pop_size, *cloud_size, population,
                cloud)) != 0) //generation.cu
            return result;
        normalize(population, pop_size); //generation.cu
        evaluate_population(THRESHOLD, PLANE_REGION, EPSILON, *cloud, *base,
                population, *cloud_size, pop_size); //fitness.cu 
        //Criterio de converxencia do algoritmo
        while (!is_converged(*population, pop_size)) {//auxiliares.cu
            if ((result = tournament_selection(*population, mating_pool,
                    pop_size, TOUR_SIZE)) != 0) //selection.cu
                    return -3;
            if ((result = recombine(mating_pool, pop_size, CROSS_PROB,
                    CROSS_D_INDEX, upper, lower, EPSILON)) != 0) //crossover.cu
                return -4;
            mutation(pop_size, mating_pool, mut_rate, mut_d_index, upper,
                    lower); //mutation.cu
            evaluate_population(THRESHOLD, PLANE_REGION, EPSILON, *cloud, *base,
                    mating_pool, *cloud_size, pop_size); //fitness.cu
            replacement(population, *mating_pool, pop_size); //replacement.cu
            exec_count++;
            recalculate_parameters(mut_d_index, mut_rate, GENE_SIZE, exec_count,
                    MAX_EXECUTIONS, MUTATION_D_INDEX); //mutation.cu
        }
        //Unha vez converxe a poboación, eliminamos os puntos que pertencen ao 
        //plano que acabamos de atopar.
        if (population->operator[](0).points_fitted > plane_min_size) {
            eliminate(population->operator[](0), cloud_size, cloud, THRESHOLD,
                    EPSILON); //eliminate.cu
            write_solution(population->operator[](0)); //auxiliares.cu
            fail = 0;
            exec_count = 0;
        } else {
            fail++;
        }
    }
    return 0;

}

/**
 * Función auxiliar que crea a nube de puntos sintética. Engade un delta 
 * aleatorio a cada punto para xerar ruído na nube.
 * 
 * @param cloud Punteiro a host_vector<Point> que contén a nube na memoria do 
 * sistema.
 * @param cloud_size Tamaño da nube
 * @param s  Estructura Solution cos parámetros do plano que vamos a empregar 
 * para xerar puntos
 * @param size Número de puntos a xerar.
 * @param offset Punto da nube onde empezamos a escribir.
 */
void generate_cloud(thrust::host_vector<Point> *cloud, Solution s, size_t size,
        size_t offset) {

    float z = 0;
    float delta = 0;
    for (int i = 0; i < 100; i++) {
        for (int j = 0; j < size / 100; j++) {
            z = ((-s.chromosome[3])+(-s.chromosome[0] * i)+
                    (-s.chromosome[1] * j)) / (s.chromosome[2]);
            delta = ((2 * ((float) rand() / (float) RAND_MAX)) - 1) / 200.0f;

            cloud->operator[](i * size / 100 + j + offset).coordinates[0] =
                    (float) i + delta;
            cloud->operator[](i * size / 100 + j + offset).coordinates[1] =
                    (float) j + delta;
            cloud->operator[](i * size / 100 + j + offset).coordinates[2] =
                    (float) z + delta;
        }
    }

}

int main() {

    srand(time(NULL));
    size_t offset = 0;
    size_t * cloud_size = (size_t *) malloc(sizeof (size_t));
    *cloud_size = CLOUD_SIZE;
    thrust::host_vector<Point> cloud(*cloud_size);
    thrust::host_vector<Solution> population(POPULATION_SIZE);
    thrust::host_vector<Solution> mating_pool(POPULATION_SIZE);
    float *upper = (float *) malloc(4 * sizeof (float));
    float *lower = (float *) malloc(4 * sizeof (float));
    Solution s;

    upper[0] = 1.0f;
    upper[1] = 1.0f;
    upper[2] = 1.0f;
    upper[3] = 1.0f;
    lower[0] = -1.0f;
    lower[1] = -1.0f;
    lower[2] = -1.0f;
    lower[3] = -1.0f;

    s.chromosome[0] = 11;
    s.chromosome[1] = 16;
    s.chromosome[2] = 14;
    s.chromosome[3] = -15;
    size_t size = 20000;
    generate_cloud(&cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = 8;
    s.chromosome[1] = 2;
    s.chromosome[2] = 3;
    s.chromosome[3] = 1;
    size = 9500;
    generate_cloud(&cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = -1;
    s.chromosome[1] = 4.5;
    s.chromosome[2] = 2.5;
    s.chromosome[3] = 12;
    size = 1200;
    generate_cloud(&cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = 18;
    s.chromosome[1] = 22;
    s.chromosome[2] = 1;
    s.chromosome[3] = 9;
    size = 10500;
    generate_cloud(&cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = -8;
    s.chromosome[1] = 0;
    s.chromosome[2] = 4;
    s.chromosome[3] = -3.25;
    size = 20000;
    generate_cloud(&cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = -5.5;
    s.chromosome[1] = 12;
    s.chromosome[2] = -1;
    s.chromosome[3] = -1;
    size = 20000;
    generate_cloud(&cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = 1;
    s.chromosome[1] = 1;
    s.chromosome[2] = 4;
    s.chromosome[3] = -5;
    size = 45000;
    generate_cloud(&cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = 12;
    s.chromosome[1] = 3;
    s.chromosome[2] = 5.5;
    s.chromosome[3] = -1;
    size = 500;
    generate_cloud(&cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = -1;
    s.chromosome[1] = 8;
    s.chromosome[2] = 2;
    s.chromosome[3] = -9;
    size = 2200;
    generate_cloud(&cloud, s, size, offset);
    offset += size;

    s.chromosome[0] = -3;
    s.chromosome[1] = -1;
    s.chromosome[2] = 8;
    s.chromosome[3] = -4;
    size = *cloud_size - offset;
    generate_cloud(&cloud, s, size, offset);
    offset += size;
    
    bucle(POPULATION_SIZE, cloud_size, MAX_FAIL, &population, &mating_pool, &cloud, SIGNIFICANT_PLANE, upper, lower);

}
